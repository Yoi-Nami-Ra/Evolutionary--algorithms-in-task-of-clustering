#include "hip/hip_runtime.h"



// Module responsible for loading data

//==============================================
//== Includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "globals.cuh"
#include "errors.cuh"
#include "distanceCalculator_Iris.cuh"
#include "dataLoader.cuh"
#include <cutil_inline.h>
#include <cutil_math.h>
#include <hip/hip_runtime.h>
//#include <shrQATest.h>

//==============================================
//== Types

//==============================================
//== Constants and Globals
static float* dDistancesVector = 0;
static float* hDistancesVector = 0;
static uint* dNeighbours = 0;
static uint* hNeighbours = 0;

//==============================================
//== Declarations
texture<float, hipTextureType1D, hipReadModeElementType> texRef;

__global__ void calculateDistances(float* vector, uint numEntries, uint blockSize, uint gridSize);
__device__ float calculateEntries(dataEntry* first, dataEntry* second);
__device__ uint vectorIdx(uint x, uint y);
__device__ float sqr(float a);
__global__ void findNeighbours( uint numEntries, uint * output );

#define BLOCK_SIZE 16
static const char* kIrisDistancesPath = "./data/iris_distances.data";

//==============================================
//== Functions
ErrorCode startCalculatingDistances() {

	ErrorCode err = LoadData();
	if (err != errOk) {
		return err;
	}

	dataStore* data = GetCurrDataStore();
	if (data == 0) {
		return GetLastErrorCode();
	}
//--------------------------------------

	// Create chanel descriptor for texture bindings
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );

	float* dData;
	// Allocate and copy data store to device memory
	uint size = data->info.numEntries * 4 * sizeof(float);
	hipMalloc( &dData, size );
	hipMemcpy( dData, data->dataVector, size, hipMemcpyHostToDevice );

	// Set texture parameters
	texRef.addressMode[ 0] = hipAddressModeWrap;
	texRef.addressMode[ 1] = hipAddressModeWrap;
	texRef.filterMode = hipFilterModeLinear;
	texRef.normalized = true;

	// Bind the array to the texture reference
	//hipBindTextureToArray( texRef, cuArray );
	uint offset = 0;
	hipBindTexture( &offset, &texRef, dData, &channelDesc, size );

	// Allocate result of transformation in device memory
	uint outputSize = data->info.numEntries * ( data->info.numEntries - 1 ) / 2;
	hipMalloc( &dDistancesVector, outputSize * sizeof(float) );

	uint hGridSize = data->info.numEntries / BLOCK_SIZE;
	while (( hGridSize * BLOCK_SIZE ) < data->info.numEntries ) {
		hGridSize++;
	}

	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE ); // thread per block
	dim3 dimGrid( hGridSize, hGridSize ); // blocks per grid

	calculateDistances<<<dimGrid, dimBlock>>>( dDistancesVector, data->info.numEntries, BLOCK_SIZE, hGridSize );

	cutilDeviceSynchronize();

	hDistancesVector = (float*)malloc( outputSize * sizeof(float) );

	if ( hDistancesVector == 0 ) {
		SetError( errNoMemory );

//		hipFreeArray( cuArray );
		hipFree( dData );
		hipFree( dDistancesVector );
		return errNoMemory;
	}

	hipMemcpy( hDistancesVector, dDistancesVector, outputSize * sizeof(float), hipMemcpyDeviceToHost );
	
	//Save results to file
	saveDistanceData();

	// no need for raw data - free it
	hipFree( dData );

	// now bind distances to texture, so we could use it for neighbours
	hipBindTexture( &offset, &texRef, dDistancesVector, &channelDesc, outputSize * sizeof(float) );

	dim3 dimBlock2( BLOCK_SIZE ); // thread per block
	dim3 dimGrid2( hGridSize ); // blocks per grid

	hipMalloc( &dNeighbours, data->info.numEntries * MAX_NEIGHBOURS * sizeof(uint) );
	findNeighbours<<<dimGrid2, dimBlock2>>>( data->info.numEntries, dNeighbours );

	cutilDeviceSynchronize();

	hipFree( dDistancesVector );
	hNeighbours = (uint*)malloc( data->info.numEntries * MAX_NEIGHBOURS * sizeof(uint) );

	if ( hNeighbours == 0 ) {
		SetError( errNoMemory );

		hipFree( dNeighbours );
		return errNoMemory;
	}

	hipMemcpy( hNeighbours, dNeighbours, data->info.numEntries * MAX_NEIGHBOURS * sizeof(uint), hipMemcpyDeviceToHost );

	hipFree( dNeighbours );

	releaseDataStore();

	return err;
}
//==============================================
__global__ void calculateDistances( float* vector, uint numEntries, uint blockSize, uint gridSize ) {
	// global position of "first" thread
	uint firstCol = blockIdx.x * BLOCK_SIZE;
	uint col = firstCol + threadIdx.x;
	uint firstRow = blockIdx.y * BLOCK_SIZE;
	uint row = firstRow + threadIdx.y;
	bool boundryBlock = false;

	__shared__ dataEntry rowData[ BLOCK_SIZE];
	__shared__ dataEntry colData[ BLOCK_SIZE];

	// Check if this isn't external block
	if ( firstRow >= firstCol && col < numEntries && row < numEntries ) {
		// Check if we should care for loading colums here
		if ( threadIdx.y == 0 ) {
			// load columns
			colData[ threadIdx.x].a = tex1Dfetch( texRef, col*4+0 );
			colData[ threadIdx.x].b = tex1Dfetch( texRef, col*4+1 );
			colData[ threadIdx.x].c = tex1Dfetch( texRef, col*4+2 );
			colData[ threadIdx.x].d = tex1Dfetch( texRef, col*4+3 );
		}
		if ( row == col ) {
			boundryBlock = true;
			// don't load rows here
		} 

		// Check if we should care for loading rows
		if ( threadIdx.x == 0 && !boundryBlock ) {
			// load rows as wel
			rowData[ threadIdx.y].a = tex1Dfetch( texRef, row*4+0 );
			rowData[ threadIdx.y].b = tex1Dfetch( texRef, row*4+1 );
			rowData[ threadIdx.y].c = tex1Dfetch( texRef, row*4+2 );
			rowData[ threadIdx.y].d = tex1Dfetch( texRef, row*4+3 );
		}

		// Sync up threads
		__syncthreads();
		// And do some calculations

		// something to do for us here ?
		if ( row > col /*&& row == 1 && col == 0*/) {
			float distance = 0;
			if (boundryBlock) {
				distance = calculateEntries(&colData[threadIdx.x], &colData[threadIdx.y]);
			} else {
				distance = calculateEntries(&colData[threadIdx.x], &rowData[threadIdx.y]);
			}
			vector[vectorIdx(col, row)] = distance;
		} else {
			vector[vectorIdx( row, col )] = 2.0f;
		}
	}
}
//==============================================

__device__ float calculateEntries(dataEntry* first, dataEntry* second) {
	float result = 0;

	result = sqr(first->a - second->a);
	result += sqr(first->b - second->b);
	result += sqr(first->c - second->c);
	result += sqr(first->d - second->d);
	result = sqrt(result);

	return result;
}
//==============================================

__device__ uint vectorIdx(uint x, uint y) {
	if ( y > x ) {
		return y * (y - 1) / 2 + x;
	} else {		
		return x * (x - 1) / 2 + y;
	}
	
}
//==============================================

__device__ float sqr(float a) {
	return a * a;
}
//==============================================

const float* getDistances() {
	return hDistancesVector;
}
//==============================================

ErrorCode releaseDistances() {
	if ( hDistancesVector ) {
		free ( hDistancesVector );
	}

	return errOk;
}
//==============================================

__global__ void findNeighbours( uint numEntries, uint * output ) {
	uint neighbours[ MAX_NEIGHBOURS];
	float neighboursDistances[ MAX_NEIGHBOURS];
	float distance = 0;;
	int i;

	uint record = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	for ( i = 0; i < MAX_NEIGHBOURS; i++ ) {
		neighbours[ i] = 0;
		neighboursDistances[ MAX_NEIGHBOURS] = 0;
	}
	

	// for each record in the data set
	for ( i = 0; i < numEntries; i++ ) {
		// if it's not the same		
		if ( record != i ) {
			// fetch distance
			distance = tex1Dfetch( texRef, vectorIdx( record, i ));			
			
			uint a = i;
			// for each neighbour already stored
			for ( int j = 0; j < MAX_NEIGHBOURS; j++ ) {
				// did we found proper one ?
				if ( neighboursDistances[ j] == 0 || distance < neighboursDistances[ j]) {					
					if ( neighboursDistances[ j] == 0 ) {
						// found empty entry - just save it here and break
						neighbours[ j] = a;
						neighboursDistances[ j] = distance;
						break;
					} else {
						// replace it and continue search with the one thrown out
						float d = neighboursDistances[ j];
						uint r = neighbours[ j];
						neighboursDistances[ j] = distance;
						neighbours[ j] = a;
						distance = d;
						a = r;
					}
				}
			} // for each neighbour
		}
	} // for each data entry

	// save results
	for ( i =0; i < MAX_NEIGHBOURS; i++ ) {
		output[ record * MAX_NEIGHBOURS + i] = neighbours[ i];
	}
}
//==============================================

const unsigned int* getNeighbours() {
	return hNeighbours;
}
//==============================================

ErrorCode releaseNeighbours() {
	if ( hNeighbours != 0 ) {
		free( hNeighbours );
	}
	return errOk;
}
//==============================================

ErrorCode loadDistanceData() {
	unsigned int numEntries = 0;
	unsigned int inputSize = numEntries * ( numEntries - 1 ) / 2;

	ErrorCode ret = errOk;

	FILE * file = fopen( kIrisDistancesPath, "r" );
	size_t res = 0;
	if ( file ) {
		res = fread( &numEntries, sizeof(unsigned int), 1, file );
		if ( res == 1 ) {
			if ( hDistancesVector == 0 ) {
				hDistancesVector = (float*)malloc( inputSize * sizeof(float) );
			}
			res = fread( &hDistancesVector, sizeof(float), inputSize, file );
			if ( res != inputSize ) {
				ret = errFileCorupted;
			}
		}
		fclose( file );
	}

	return ret;
}
//==============================================

errorCode saveDistanceData() {
	// check if we have something worh to save
	if ( hDistancesVector == 0 ) {
		return errNoData;
	}

	unsigned int numEntries = numEntries();
	unsigned int outputSize = numEntries * ( numEntries - 1 ) / 2;

	FILE * file = fopen( kIrisDistancesPath, "w" );
	size_t res = 0;
	if ( file ) {
		res = fwrite( &( numEntries ), sizeof(unsigned int), 1, file );
		if (res == 1) {
			res = fwrite( hDistancesVector, sizeof(float), outputSize, file );
		}
		if ( res!=1 && res!=outputSize ) {
			err = errFileWrite;
		}

		fclose( file );
	} else {
		return errFileWrite;
	}

	return errOk;
}
//==============================================