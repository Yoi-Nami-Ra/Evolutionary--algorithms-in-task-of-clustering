#include "hip/hip_runtime.h"



//====================================================================
//== Includes
#include <stdlib.h>
#include <stdio.h>
//#include <string.h>
//#include <math.h>
#include "globals.cuh"
#include "errors.cuh"
#include "clustering.cuh"
#include "distanceCalculator.cuh"
#include "dataLoader.cuh"
#include <cutil_inline.h>
#include <cutil_math.h>
#include <hip/hip_runtime.h>
#include <time.h>

//====================================================================
//== Types

//====================================================================
//== Constants and Globals
static float * dDistances;
static float * dNeighbours;
static unit * hPopulationPool;
static unit * dPopulationPool;
static unsigned int populationSize;

const char threadsPerBlock = 50;

//====================================================================
//== Declarations
texture<float, hipTextureType1D, hipReadModeElementType> texRefDistances;
texture<float, hipTextureType1D, hipReadModeElementType> texRefNeighbour;

// host
ErrorCode runAlgorithms( unsigned int steps );
// device
__global__ void randomPopulation( unsigned int popSize, unsigned char blockSize, unit * population, unsigned int numEntries );
__global__ void kernelMembershipAndDensity( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution );
__device__ float distance( unsigned int a, unsigned int b );
__device__ uint distanceIdx(uint x, uint y);
__global__ void kernelConnectivity( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution );
__global__ void kernelDisconnectivity( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution );
__global__ void kernelCorectness( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution );
__global__ void kernelSorting( float * fitnesResults, bool * dominanceMatrix, 
	unsigned int blocksPerSolution, unsigned int populationSize );
__global__ void kernelDominanceCount( bool * dominanceMatrix, unsigned int * dominanceCounts, unsigned int popSize );
__global__ void kernelFrontDensity( unsigned int * front, unsigned int frontSize, unsigned int blocksPerSolution,
	float * fitnesResults, float * frontDensities );

//====================================================================
//== Functions

ErrorCode generateRandomPopulation( unsigned int popSize ) {

	populationSize = popSize;
	hPopulationPool = (unit*)malloc( popSize * sizeof(unit) );
	srand( time( 0 ));

	for ( int k = 0; k < popSize; k++ ) {
		// attributes
		hPopulationPool[k].attr.clusterMaxSize = rand() % MAX_CLUSTER_SIZE + 1;
		hPopulationPool[k].attr.numNeighbours = rand() % MAX_NEIGHBORS + 1;
		unsigned int clustersSum = MEDOID_VECTOR_SIZE;
		unsigned int proposal;
		bool proposalOk = false;
		for ( int i = 0; i < MEDOID_VECTOR_SIZE; i++ ) {
			do {
				proposalOk = true;
				proposal = rand() % numEntries();
				for ( int j = 0; j < i; j++ ) {
					if ( proposal == hPopulationPool[k].medoids[ j] ) {
						proposalOk = false;
						break;
					}
				}
			} while ( !proposalOk );
			hPopulationPool[k].medoids[ i] = proposal;

			if ( clustersSum > 0 ) {
				proposal = rand() % hPopulationPool[k].attr.clusterMaxSize + 1;
				if ( clustersSum < proposal ) {
					proposal += clustersSum - proposal;
				}
				clustersSum -= proposal;
				hPopulationPool[k].clusters[ i] = proposal;
			} else {
				hPopulationPool[k].clusters[ i] = 0;
			}
		} // for each medoid in vector
	} // for each member of population

	hipMalloc( &dPopulationPool, popSize * sizeof(unit) );
	hipMemcpy( dPopulationPool, hPopulationPool, popSize * sizeof(unit), hipMemcpyHostToDevice );

	return errOk;
}
//====================================================================

ErrorCode runClustering( unsigned int popSize, unsigned int steps ) {
	// Bind textures
	//   Chanel descriptor
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );
	//   Texture properties	
	texRefDistances.addressMode[ 0] = hipAddressModeWrap;
	texRefDistances.addressMode[ 1] = hipAddressModeWrap;
	texRefDistances.filterMode = hipFilterModeLinear;
	texRefDistances.normalized = true;
	texRefNeighbour.addressMode[ 0] = hipAddressModeWrap;
	texRefNeighbour.addressMode[ 1] = hipAddressModeWrap;
	texRefNeighbour.filterMode = hipFilterModeLinear;
	texRefNeighbour.normalized = true;

	//   Allocate memory for distances
	unsigned int offset = 0;
	
	unsigned int distancesSize = numEntries() * ( numEntries() -1 ) /2 * sizeof(float);
	hipMalloc( &dDistances, distancesSize );
	hipMemcpy( dDistances, getDistances(), distancesSize, hipMemcpyHostToDevice );
	//   bind distances to texture
	hipBindTexture( &offset, &texRefDistances, dDistances, &channelDesc, distancesSize );

	//   Allocate memory for distances	
	unsigned int neighbourSize = numEntries() * MAX_NEIGHBORS * sizeof(unsigned int);
	hipMalloc( &dNeighbours, neighbourSize );
	hipMemcpy( dNeighbours, getNeighbours(), neighbourSize, hipMemcpyHostToDevice );
	//   bind distances to texture
	hipBindTexture( &offset, &texRefNeighbour, dNeighbours, &channelDesc, neighbourSize );

	// generate startup population
	generateRandomPopulation( popSize );

	runAlgorithms( steps );

	return errOk;
}
//====================================================================

__global__ void randomPopulation( unsigned int popSize, unsigned char blockSize, unit * population, unsigned int numEntries ) {
	unsigned int index = blockIdx.x * blockSize + threadIdx.x;

	
}
//====================================================================

ErrorCode runAlgorithms( unsigned int steps ) {
	// Set ups
	float * hFitnesResults = (float*)malloc( populationSize * 4 * sizeof(float) );
	float * dFitnesResults = 0;
	char * dMembership = 0;
	unsigned int blocks = numEntries() / threadsPerBlock;
	while ( blocks * threadsPerBlock < numEntries()) {
		blocks++;
	}

	// populationSize x numObjectives x blocksPerSolution
	hipMalloc( &dFitnesResults, populationSize * 4 * blocks * sizeof(float) );
	hipMalloc( &dMembership, populationSize * numEntries() * sizeof(char) );

	// population dominations
	bool * dDominanceMatrix = 0;
	unsigned int * dDominanceCounts = 0;
	hipMalloc( &dDominanceMatrix, populationSize * populationSize * sizeof(bool) );
	hipMalloc( &dDominanceCounts, populationSize * sizeof(unsigned int) );

	dim3 dimGrid( blocks, populationSize );
	dim3 dimBlock( threadsPerBlock );

	dim3 dimGrid2( populationSize );
	dim3 dimBlock2( MEDOID_VECTOR_SIZE );

	unsigned int solutionsLeft = 0;
	bool * solutionsSelected = (bool*)malloc( populationSize * sizeof(bool) );
	unsigned int * solutionFronts = (unsigned int*)malloc( populationSize * ( populationSize + 1 ) * sizeof(unsigned int));
	unsigned int currFront;
	unsigned int currFrontSize;

	bool * hDominanceMatrix = (bool*)malloc( populationSize * populationSize * sizeof(bool));
	unsigned int * hDominanceCounts = (unsigned int*)malloc( populationSize * sizeof(unsigned int));
	float * dFrontDensities = 0;
	hipMalloc( &dFrontDensities, populationSize * sizeof(float) );
	float * hFrontDensities = (float*)malloc( populationSize * sizeof(float) );


	for (int i = 0; i < steps; i++ ) {
		// membership and density phase
		kernelMembershipAndDensity<<<dimGrid, dimBlock>>>( dFitnesResults, dMembership, threadsPerBlock, dPopulationPool, numEntries(), blocks );
		cutilDeviceSynchronize();

		// connectivity phase
		kernelConnectivity<<<dimGrid, dimBlock>>>( dFitnesResults, dMembership, threadsPerBlock, dPopulationPool, numEntries(), blocks );
		cutilDeviceSynchronize();

		// sum up results for density and Connectivity
		// TODO:

		// disconnectivity phase
		kernelDisconnectivity<<<dimGrid2, dimBlock2>>>( dFitnesResults, dMembership, MEDOID_VECTOR_SIZE, dPopulationPool, numEntries(), populationSize );
		cutilDeviceSynchronize();

		// correctness phase
		kernelCorectness<<<dimGrid2, dimBlock2>>>( dFitnesResults, dMembership, MEDOID_VECTOR_SIZE, dPopulationPool, numEntries(), populationSize );
		cutilDeviceSynchronize();

		// sorting
		kernelSorting<<<populationSize, populationSize>>>( dFitnesResults, dDominanceMatrix, blocks, populationSize );
		cutilDeviceSynchronize();

		kernelDominanceCount<<<1, populationSize>>>( dDominanceMatrix, dDominanceCounts, populationSize );
		cutilDeviceSynchronize();

		hipMemcpy( hDominanceMatrix, dDominanceMatrix, populationSize * populationSize * sizeof(bool), hipMemcpyDeviceToHost );
		hipMemcpy( hDominanceCounts, dDominanceCounts, populationSize * sizeof( unsigned int ), hipMemcpyDeviceToHost );

		// setup fronts
		solutionsLeft = populationSize;
		currFront = 0;

		int j;
		for ( j = 0; j < populationSize; j++ ) {
			solutionsSelected[ j] = false;
		}

		// front grouping phase
		while ( solutionsLeft > 0 ) {
			currFrontSize = 0;
			// select solutions for current front - where domination count is 0
			for ( j = 0; j < populationSize; j++ ) {
				if ( !solutionsSelected[ j] && hDominanceCounts[ j] == 0 ) {
					solutionFronts[ currFront * populationSize + (++currFrontSize)] = j;
					solutionsSelected[ j] = true;
					solutionsLeft--;
				}
			}
			solutionFronts[ currFront * populationSize] = currFrontSize;
			solutionsLeft -= currFrontSize;
			
			if ( solutionsLeft > 0 ) {
				// for each solution dominated by solution from this front - reduce domination count
				for ( j = 0; j < currFrontSize; j++ ) {
					for ( int k = 0; k < populationSize; k++ ) {
						if ( hDominanceMatrix[ solutionFronts[ currFront * populationSize + j + 1] * populationSize + k] ) {
							hDominanceCounts[ k] -= 1;
						}
					}
				}
			}

			// now for next front
			currFront++;
		}

		// selection
		solutionsLeft = populationSize / 2; // select half size of population
		for ( j = 0; j < populationSize; j++ ) {
			solutionsSelected[ j] = false;
		}

		currFront = 0;
		while ( solutionsLeft > 0 ) {
			// if we need more than the current front can offer
			if ( solutionsLeft > solutionFronts[ currFront * populationSize] ) {
				for ( j = 0; j < solutionsLeft > solutionFronts[ currFront * populationSize]; j++ ) {
					solutionsSelected[ solutionFronts[ currFront * populationSize + j + 1]] = true;
				}
			} else {
				// this front has more than we need
				unsigned int currFrontSize = solutionFronts[ currFront * populationSize];

				// Calculate densities for solutions in this front
				kernelFrontDensity<<<currFrontSize, 4>>>( &solutionFronts[ currFront * populationSize + 1],
					currFrontSize, blocks, dFitnesResults, dFrontDensities );
				cutilDeviceSynchronize();

				// Export results to Host
				hipMemcpy( hFrontDensities, dFrontDensities, currFrontSize * sizeof(float), hipMemcpyDeviceToHost );
				
				bool * thisFronSelection = (bool*)malloc( populationSize * sizeof(bool));
				unsigned int smallest = 0;

				// Select first selectionLeft solutions and find the smallest one (bug density)
				for ( j = 0; j < currFrontSize; j++ ) {
					thisFronSelection [ j] = ( j < solutionsLeft );
					if ( thisFronSelection[ j] ) {
						if ( hFrontDensities[ j] != -1 && ( hFrontDensities[ j] < hFrontDensities[ smallest] || hFrontDensities[ smallest] == -1 ) ) {
							smallest = j;
						}
					}
				} // for j

				// Now for each solution not selected at first, check if it's bigger than the smallest
				// If so, replece it with smallest
				if  ( hFrontDensities[ smallest] != -1 ) {
					for (; j < solutionFronts[ currFront * populationSize]; j++ ) {
						if ( hFrontDensities[ j] == -1 || hFrontDensities[ j] > hFrontDensities[ smallest] ) {
							thisFrontSelection[ smallest] = false;
							thisFrontSelection[ j] = true;
							smallest = j;
							for ( int k = 0; k < j; k++ ) {
								if ( thisFronSelection[ k] ) {
									if ( hFrontDensities[ k] != -1 && ( hFrontDensities[ k] < hFrontDensities[ smallest] || hFrontDensities[ smallest] == -1 ) ) {
										smallest = k;
									}
								}
							} // for k
						}
					} // for j
				}

				// now mark solutions in main selection table
				for ( j = 0; j < currFrontSize; j++ ) {
					if ( thisFrontSelection[ j] ) {
						solutionsSelected[ solutionFronts[ currFront * populationSize + j + 1]] = true;
						solutionsLeft--;
					}
				}// for j
			}

			currFront++;
		} // while

		// crossing
		unsigned int halfPopulation = populationSize / 2;
		unsigned int * hBreedingTable = (unsigned int*)malloc( halfPopulation * 3 );
		unsigned int currParent1 = 0;
		unsigned int currParent2 = 0;
		unsigned int currChild = 0;

		for ( j = 0; j < populationSize; j++ ) {
			if ( solutionsSelected[ j] ) {
				// place for parent
				if ( currParent1 <= currParent2 ) {
					// place taken by first parent
					hBreadingTable[ ( currParent1++ ) * 3] = j;
					hBreadingTable[ ( currParent1++ ) * 3 + 1] = j;
				} else {
					hBreadingTable[ ( currParent2++ ) * 3 + 1] = j;
					hBreadingTable[ ( currParent2++ ) * 3 ] = j;
				}
			} else {
				// place for child
				hBreadingTable[ ( currParent2++ ) * 3 + 2] = j;
			}
		}		
	}

	return errOk;
}
//====================================================================

__global__ void kernelMembershipAndDensity( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution ) {
	unsigned int solution = blockIdx.y;
	unsigned int rekord = blockIdx.x * threadsPerBlock + threadIdx.x;

	__shared__ unit thisSolution;
	__shared__ float density[ 256]; // shared table to hold density results for futher calculation

	if ( threadIdx.x == 0 ) {
		// first thread should load solution for this block to shared memory
		thisSolution = populationPool[ solution];
	}

	// Sync up threads
	__syncthreads();

	float prevDistance = distance( rekord, thisSolution.medoids[ 0] );
	float currDistance;
	unsigned int res = 0;

	for ( int i = 1; i < MEDOID_VECTOR_SIZE; i++ ) {
		currDistance = distance ( rekord, thisSolution.medoids[ 0] );
		if ( currDistance > prevDistance ) {
			prevDistance = currDistance;
			res = i;
		}
	}

	density[ threadIdx.x] = prevDistance;

	membership[ solution * numEntries + rekord] = res;

	// Sync up threads
	__syncthreads();	
	if ( threadIdx.x == 0 ) {
		// sum solutions from all threads in this block
		currDistance = 0;
		for ( int i = 0; i < threadsPerBlock; i++ ) {
			currDistance += density[ i];
		}

		// sum all solutions for this block
		fitnesResults[ solution * 4 * blocksPerSolution + 0 * blocksPerSolution + blockIdx.x] = currDistance;
	}

	// Sync once more
	__syncthreads();

	// now sum all results for this solution (from each block)
	if ( blockIdx.x == 0 && threadIdx.x == 0 ) {
		currDistance = 0;
		for ( int i = 0; i < blocksPerSolution; i++ ) {
			currDistance += fitnesResults[ solution * 4 * blocksPerSolution + 0 * blocksPerSolution + i];
		}
		fitnesResults[ solution * 4 * blocksPerSolution + 0 * blocksPerSolution + 0] = currDistance;
	}
}
//====================================================================

__device__ float distance( unsigned int a, unsigned int b ) {
	return tex1Dfetch( texRefDistances, distanceIdx( a, b ));
}
//====================================================================

__device__ uint distanceIdx(uint x, uint y) {
	if ( y > x ) {
		return y * (y - 1) / 2 + x;
	} else {		
		return x * (x - 1) / 2 + y;
	}
}
//====================================================================

__device__ unsigned int neighbour( unsigned int record, unsigned int num ) {
	return tex1Dfetch( texRefNeighbour, record * MAX_NEIGHBORS + num );
}
//====================================================================

__global__ void kernelConnectivity( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution ) {
	
	unsigned int solution = blockIdx.y;
	unsigned int record = blockIdx.x * threadsPerBlock + threadIdx.x;
	unit thisSolution = populationPool[ solution];
	unsigned int memberOf = membership[ solution * numEntries + record];
	unsigned int numOfNeighbours = thisSolution.attr.numNeighbours;

	__shared__ float blockResults[256];
	float result = 0;

	for ( int i = 0; i < numOfNeighbours; i++ ) {
		if ( memberOf == membership[ solution * numEntries + i] ) {
			result += 1.0f / (float)numOfNeighbours;
		}
	}

	blockResults[ threadIdx.x] = result;

	// Sync and sum solutions for this block
	__syncthreads();
	if ( threadIdx.x == 0 ) {
		// sum solutions from all threads in this block
		result = 0;
		for ( int i = 0; i < threadsPerBlock; i++ ) {
			result += blockResults[ i];
		}

		// sum all solutions for this block
		fitnesResults[ solution * 4 * blocksPerSolution + 1 * blocksPerSolution + blockIdx.x] = result;
	}

	// Sync once more
	__syncthreads();

	// now sum all results for this solution (from each block)
	if ( blockIdx.x == 0 && threadIdx.x == 0 ) {
		result = 0;
		for ( int i = 0; i < blocksPerSolution; i++ ) {
			result += fitnesResults[ solution * 4 * blocksPerSolution + 1 * blocksPerSolution + i];
		}
		fitnesResults[ solution * 4 * blocksPerSolution + 1 * blocksPerSolution + 0] = result;
	}

}
//====================================================================

__global__ void kernelDisconnectivity( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution ) {

	__shared__ unsigned int clusters[ MEDOID_VECTOR_SIZE];
	__shared__ unsigned int comparisions[ MEDOID_VECTOR_SIZE];
	__shared__ unsigned int counts[ MEDOID_VECTOR_SIZE];


	// label medoids to their clusters
	if ( threadIdx.x == 0 ) {
		for ( int i=0, j=0; i<MEDOID_VECTOR_SIZE && j<MEDOID_VECTOR_SIZE; i++ ) {
			for (int k=0; k<populationPool[ blockIdx.y].clusters[i]; k++,j++) {
				clusters[ j] = i;
			}
		}
	}

	__syncthreads();

	float currDistance  = 0;
	comparisions[ threadIdx.x] = 0;
	counts[ threadIdx.x] = 0;
	// For each medoid in the vector
	for ( unsigned int i=0; i < MEDOID_VECTOR_SIZE; i++ ) {
		if ( i == threadIdx.x || clusters[ threadIdx.x] == clusters[ i] ) {
			// if medoid the same or same cluster - skip
			continue;
		}
		comparisions[ threadIdx.x]++;
		currDistance = distance( populationPool[ blockIdx.y].medoids[ threadIdx.x], populationPool[ blockIdx.y].medoids[ i] );
		// now find MND for this medoid
		for ( unsigned int j = 0; j < MEDOID_VECTOR_SIZE; j++ ) {
			if ( j == i || j == threadIdx.x ) {
				continue;
			}
			// counts if someone else is closer
			if ( distance( populationPool[ blockIdx.y].medoids[ threadIdx.x], populationPool[ blockIdx.y].medoids[ j] ) < currDistance ) {
				counts[ threadIdx.x]++;
			}
			if ( distance( i, j ) < currDistance ) {
				counts[ threadIdx.x]++;
			}
		}
	}

	__syncthreads();

	if ( threadIdx.x == 0 ) {
		float compars = 0;
		float count = 0;
		for ( int i = 0; i < MEDOID_VECTOR_SIZE; i++ ) {
			compars += comparisions[ i];
			count += counts[ i];
		}
		fitnesResults[ blockIdx.x * 4 * blocksPerSolution + 2 * blocksPerSolution + 0] = count/compars;
	}
}
//====================================================================

__global__ void kernelCorectness( float * fitnesResults, char * membership,
	unsigned int threadsPerBlock, unit * populationPool, unsigned int numEntries, unsigned int blocksPerSolution ) {

	__shared__ char checks[ MEDOID_VECTOR_SIZE];
	__shared__ unsigned int medoids[ MEDOID_VECTOR_SIZE];

	if ( threadIdx.x == 0 ) {
		memcpy( medoids, populationPool[ blockIdx.y].medoids, MEDOID_VECTOR_SIZE * sizeof(unsigned int) );
	}

	__syncthreads();

	unsigned int thisMedoid = medoids[ threadIdx.x];
	char count = 0;
	
	for ( int i=0; i < MEDOID_VECTOR_SIZE; i++ ) {
		if (i == threadIdx.x) {
			continue;
		}

		if ( thisMedoid == medoids[ i] ) {
			count++;
		}
	}

	checks[ threadIdx.x] = count;

	__syncthreads();

	if ( threadIdx.x == 0 )  {
		for ( int i = 0; i < threadsPerBlock; i++ ) {
			count += checks[ i];
		}
		fitnesResults[ blockIdx.x * 4 * blocksPerSolution + 3 * blocksPerSolution + 0] = count;
	}
}
//====================================================================

// <<< populationSize, populationSize >>>
__global__ void kernelSorting( float * fitnesResults, bool * dominanceMatrix, 
	unsigned int blocksPerSolution, unsigned int populationSize ) {

	__shared__ bool dominating[ MAX_POPULATION_SIZE];
	__shared__ float thisSolutionFitnesResults[4];

	if ( threadIdx.x == 0 ) {
		thisSolutionFitnesResults[ 0] = fitnesResults[ blockIdx.x * 4 * blocksPerSolution + 0 * blocksPerSolution + 0];
		thisSolutionFitnesResults[ 1] = fitnesResults[ blockIdx.x * 4 * blocksPerSolution + 1 * blocksPerSolution + 0];
		thisSolutionFitnesResults[ 2] = fitnesResults[ blockIdx.x * 4 * blocksPerSolution + 2 * blocksPerSolution + 0];
		thisSolutionFitnesResults[ 3] = fitnesResults[ blockIdx.x * 4 * blocksPerSolution + 3 * blocksPerSolution + 0];
	}

	__syncthreads();

	dominating[ threadIdx.x] = true;
	for ( int i = 0; i < 4 ;i++ ) {
		if ( fitnesResults[ threadIdx.x * 4 * blocksPerSolution + i * blocksPerSolution + 0] >=
			thisSolutionFitnesResults[ 0] ) {
				dominating[ threadIdx.x] = false;
		}
	}

	__syncthreads();

	if ( threadIdx.x == 0 ) {
		memcpy( &dominanceMatrix[ blockIdx.x * populationSize], dominating, populationSize * sizeof(float) );
	}
}
//====================================================================

// <<< 1, ppulationSize >>>
__global__ void kernelDominanceCount( bool * dominanceMatrix, unsigned int * dominanceCounts, unsigned int popSize ) {

	unsigned int count = 0;

	for ( int i = 0; i < popSize; i++ ) {
		count += dominanceMatrix[ i * popSize + threadIdx.x];
	}

	dominanceCounts[ threadIdx.x] = count;
}
//====================================================================

// <<< numSolutions, kryterions >>>
__global__ void kernelFrontDensity( unsigned int * front, unsigned int frontSize, unsigned int blocksPerSolution,
	float * fitnesResults, float * frontDensities ) {

	__shared__ float solutionDensities [ 4];

	unsigned int lesser;
	bool lesserFound = false;
	float lesserResult;
	unsigned int bigger;
	bool biggerFound = false;
	float biggerResult;

	float thisResult = fitnesResults[ front[ blockIdx.x] * 4 * blocksPerSolution + threadIdx.x * blocksPerSolution + 0];
	float currResult;

	for ( int i = 0; i < frontSize; i++ ) {
		if ( threadIdx.x == i ) {
			// skip if same
			continue;
		}

		currResult = fitnesResults[ front[ i] * 4 * blocksPerSolution + threadIdx.x * blocksPerSolution + 0];
		// check if lesser
		if ( thisResult > currResult ) {
			if ( !lesserFound ) {
				lesser = i;
				lesserFound = true;
				lesserResult = currResult;
			} else {
				if ( lesserResult < currResult ) {
					lesser = i;
					lesserFound = true;
					lesserResult = currResult;
				}
			}
		}

		// check if bigger
		if ( thisResult < currResult ) {
			if ( !biggerFound ) {
				bigger = i;
				biggerFound = true;
				biggerResult = currResult;
			} else {
				if ( biggerResult > currResult ) {
					bigger = i;
					biggerResult = currResult;
				}
			}
		}
	} // for each solution in this front

	// is this edge solution ?
	if ( !lesserFound || !biggerFound ) {
		frontDensities[ threadIdx.x] = -1;  
	} else {
		frontDensities[ threadIdx.x] = biggerResult - lesserResult;
	}

	if ( threadIdx.x == 0 ) {
		for ( int i = 1; i < 4; i++ ) {
			frontDensities[ 0] = frontDensities[ i];
		}
		frontDensities[ blockIdx.x] = solutionDensities[ 0];
	}
}
//====================================================================

// <<< 1, popSize/2 >>>
__global__ void kernelCrossing( unsigned int popSize, unit * population, unsigned int * breedingTable ) {
	__shared__ bool crossTemplate[ MEDOID_VECTOR_SIZE];

	unsigned int stepSize = MEDOID_VECTOR_SIZE / CROS_FACTOR;
	bool mark = true;
	if ( threadIdx.x == 0 ) {		
		for ( int i = 0, j = 0; i < MEDOID_VECTOR_SIZE; i++ ) {
			if ( j => stepSize ) mark = !mark;
			crossTemplate[ i] = mark;
		}
	}

	__syncthreads();

	char parent1Clusters[ MEDOID_VECTOR_SIZE];
	char parent2Clusters[ MEDOID_VECTOR_SIZE];
	unsigned int parent1 = breedingTable[ threadIdx.x * 3];
	unsigned int parent2 = breedingTable[ threadIdx.x * 3 + 1];
	unsigned int child = breedingTable[ threadIdx.x * 3 + 2];


	unsigned int index = 0;
	cluster = 0;
	for ( int i = 0, index = 0; i < MEDOID_VECTOR_SIZE && index < MEDOID_VECTOR_SIZE; i++ ) {
		for ( int j = 0; j < population[ parent1Clusters].clusters[ i]; j++ ) {
			parent1Cluster[ index++] = cluster;
		}
		cluster++;
	}

	index = 0;
	cluster = 0;
	for ( int i = 0, index = 0; i < MEDOID_VECTOR_SIZE && index < MEDOID_VECTOR_SIZE; i++ ) {
		for ( int j = 0; j < population[ parent2Cluster].clusters[ i]; j++ ) {
			parent2Cluster[ index++] = cluster;
		}
		cluster++;
	}

	char childrenClusters[ MEDOID_VECTOR_SIZE];

	for ( int i = 0; i < MEDOID_VECTOR_SIZE; i++ ) {
		if ( crossTemplate[ i] ) {
			childrenCluster[ i] = parent1Clusters[ i];
			population[ child].medoids[i] = population[ parent1].medoids[ i];
		} else {
			childrenCluster[ i] = parent2Clusters[ i];
			population[ child].medoids[i] = population[ parent2].medoids[ i];
		}
	}
	population[ child].attr.clusterMaxSize = population[ parent1].attr.clusterMaxSize;
	population[ child].attr.numNeighbours = population[ parent1].attr.numNeighbours;
}