#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <cutil_inline.h>
#include <shrQATest.h>

#include "dataLoader_Iris.cuh"
#include "dataLoader_Test.cuh"
#include "dataLoader_Wine.cuh"
#include "dataLoader_Cancer.cuh"
#include "distanceCalculator.cuh"
#include "clustering.cuh"
#include "errors.cuh"
#include <time.h>

#define kReportsFileName "_CUDAReportsFile.txt"
#define kReportsFileNameXls "_CUDAReportsFileXls.txt"

// Host code
int main(int argc, char** argv)
{
    int devID;	
	hipDeviceProp_t props;
	
    shrQAStart(argc, argv);

	//Check which GPU is used
	cutilChooseCudaDevice(argc, argv);
	
	//Get GPU information
	cutilSafeCall(hipGetDevice(&devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
	printf("Device %d: \"%s\" with Compute %d.%d capability\n", 
			devID, props.name, props.major, props.minor);

	ErrorCode err = errOk;	

	algResults results;
	DataStore dataStore;

	//= Setup loaders
	//--------------------
	SetupIrisLoader();
	SetupTestLoader();
	SetupWineLoader();
	SetupCancerLoader();
	
	//= Load data
	//--------------------
	err = GetCalculatedDistances( 3, &dataStore );

	unsigned int cNeighbours = 0;
    unsigned int cClusters = 0;
    unsigned int cMedoids = 0;
    unsigned int cPopSize =  0;
    unsigned int cSteps = 0;
    unsigned int cRepeat =  0;

	unsigned int stepsMedoids;
	unsigned int stepsClusters;
	unsigned int stepsNeighbours;
		
	/*
	 
    char stateSaved = 1;
    unsigned int sPopSize = 256;
    unsigned int sSteps = 502;
	*/
	char stateSaved = 0;
    unsigned int sPopSize = 0;
    unsigned int sSteps = 0;

	FILE * reportsFile =  NULL;
	char * reportsFileName;
	char * xlsReportsFileName;
	unsigned int fileNameLength = strlen( kReportsFileName ) + strlen( dataStore.info.name );
	reportsFileName = (char*)malloc( fileNameLength + 1 );
	sprintf( reportsFileName, "%s%s", dataStore.info.name, kReportsFileName );
	fileNameLength = strlen( kReportsFileNameXls ) + strlen( dataStore.info.name );
	xlsReportsFileName = (char*)malloc( fileNameLength + 1 );
	sprintf( xlsReportsFileName, "%s%s", dataStore.info.name, kReportsFileNameXls );
	
	if ( reportsFile == NULL && !stateSaved ) {
		reportsFile = fopen( reportsFileName, "w" );
	}

	if (reportsFile != NULL ) {
		fclose( reportsFile );
		reportsFile = NULL;
	}

	if ( !stateSaved ) {
		FILE * xlsReportFile = fopen( xlsReportsFileName, "w" );
		if ( xlsReportFile != NULL ) {
			fclose( xlsReportFile );
			xlsReportFile = NULL;
		}
	}


	// calculate how big changes per step
	stepsMedoids = ( ( dataStore.info.numEntries / 4 - 3 ) / 3 );
	if ( stepsMedoids == 0 ) stepsMedoids = 1;
	
	stepsNeighbours = ( kMaxNeighbours - 1 ) / 2;
	if ( stepsNeighbours == 0 ) stepsNeighbours = 1;

	time_t currTime = 0;
	float timeDiff = 0.0;


	if ( err == errOk ) {
		for ( cPopSize = 4; cPopSize <= 256; cPopSize *= 4 ) { // 4 - 16 - 64 - 256
			// now the evolution params                    
            for ( cSteps = 2; cSteps <= 1002; cSteps += 500 ) { // 2 - 502 - 1002
				if ( stateSaved ) {
                    cPopSize = sPopSize;
                    cSteps = sSteps;
                    stateSaved = 0;
                }

				if ( reportsFile == NULL ) {
					reportsFile = fopen( reportsFileName, "a" );
				}

				if (reportsFile != NULL ) {
					fprintf( reportsFile, "---------------------------------\n" );
					printf( "---------------------------------\n" );
					fprintf( reportsFile, " medoids: %d clusters: %d neighbours: %d\n", MEDOID_VECTOR_SIZE, MAX_CLUSTER_SIZE, kMaxNeighboursToUSe );
					printf( " medoids: %d clusters: %d neighbours: %d\n", MEDOID_VECTOR_SIZE, MAX_CLUSTER_SIZE, kMaxNeighboursToUSe );
					fprintf( reportsFile, " popSize: %d steps: %d\n", cPopSize, cSteps );
					printf( " popSize: %d steps: %d\n", cPopSize, cSteps );
					fprintf( reportsFile, " Results:\n" );
					printf( " Results:\n" );
					fclose( reportsFile );
					reportsFile = NULL;
				}
				CleanAlgResults( results );
                for ( cRepeat = 0; cRepeat < 5; cRepeat++ ) {
                    time( &currTime );
					err = runClustering( cPopSize, cSteps, &dataStore, &results );
					timeDiff = difftime( time( NULL ), currTime );

					if ( results.time.min == 0 || results.time.min > timeDiff ) {
						results.time.min = timeDiff;
					}
					if ( results.time.max == 0 || results.time.max < timeDiff ) {
						results.time.max = timeDiff;
					}
					results.time.sum += timeDiff;
                                
                    if ( err != errOk ) {
                        break;
                    }
                }
				results.time.mean = results.time.sum / 5.0;
				printf( "=============================================\n" );
                if ( err != errOk ) {
                    break;
                }
                            
				if ( reportsFile == NULL ) {
					reportsFile = fopen( reportsFileName, "a" );
				}

				if (reportsFile != NULL ) {
					fprintf( reportsFile, " BDI:  %f / %f / %f\n", results.bdi.min, results.bdi.mean, results.bdi.max );
					printf( " BDI:  %f / %f / %f\n", results.bdi.min, results.bdi.mean, results.bdi.max );
					fprintf( reportsFile, " DI:  %f / %f / %f\n", results.di.min, results.di.mean, results.di.max );
					printf( " DI:  %f / %f / %f\n", results.di.min, results.di.mean, results.di.max );
					fprintf( reportsFile, " Rand:  %f / %f / %f\n", results.rand.min, results.rand.mean, results.rand.max );
					printf( " Rand:  %f / %f / %f\n", results.rand.min, results.rand.mean, results.rand.max );
					fprintf( reportsFile, " Time:  %f / %f / %f\n\n", results.time.min, results.time.mean, results.time.max );
					printf( " Time:  %f / %f / %f\n\n", results.time.min, results.time.mean, results.time.max );
					fclose( reportsFile );
					reportsFile = NULL;

				} else {
					// Filed to write report
				}

				// xls readable file
				{
					// medoids, clusters, neighbours, popSize, Steps, BDI_min, BDI_mean, BDI_max, DI_min, DI_mean, DI_max, Rand_min, Rand_mean, Rand_max, Time_min, Time_mean, Time_max
					FILE * xlsReportFile = fopen( xlsReportsFileName, "a" );
					if ( xlsReportFile != NULL ) {
						fprintf( xlsReportFile, "%u, %u, %u, %u, %u, ",
							cMedoids, cClusters, cNeighbours, cPopSize, cSteps );
						fprintf( xlsReportFile, "%f, %f, %f, ",
							results.bdi.min, results.bdi.mean, results.bdi.max );
						fprintf( xlsReportFile, "%f, %f, %f, ",
							results.di.min, results.di.mean, results.di.max );
						fprintf( xlsReportFile, "%f, %f, %f, ",
							results.rand.min, results.rand.mean, results.rand.max );
						fprintf( xlsReportFile, "%f, %f, %f\n", results.time.min, results.time.mean, results.time.max );
						fclose( xlsReportFile );
						xlsReportFile = NULL;
					}
				}
            } // for steps
            if ( err != errOk ) {
                break;
            }
        } // pop size
	} // error
	//====
	

	shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
	return 0;
}