/**
 22/02/2012
 Jaroslaw Wojtasik

 Cuda_test

 distanceCalculator_test.cu
 **/

//============================================================================
//== Includes
#include <cutil_inline.h>
#include <cutil_math.h>
#include <hip/hip_runtime.h>
#include "testsGeneral.cuh"

//============================================================================
//== Globals
//============================================================================
//== Declarations

/*
 * Tests raw data binding to texture
 */
bool test01 ();

//============================================================================
//== Functions

void runDistancesTests() {
	prepareTests;

	makeTest( "Texture binding", test01() );

	endTests;
}
//----------------------------------------------------------------------------

bool test01 () {
	return false;
}
//----------------------------------------------------------------------------
//----------------------------------------------------------------------------
