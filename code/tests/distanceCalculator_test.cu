/**
 22/02/2012
 Jaroslaw Wojtasik

 Cuda_test

 distanceCalculator_test.cu
 **/

//============================================================================
//== Includes
#include <cutil_inline.h>
#include <cutil_math.h>
#include <hip/hip_runtime.h>
#include "testsGeneral.cuh"
#include "dataLoader.cuh"
#include "distanceCalculator.cuh"

//============================================================================
//== Globals
//============================================================================
//== Declarations

/*
 * Tests raw data binding to texture
 */
bool test01 ();

bool test02 ();

//============================================================================
//== Functions

void runDistancesTests() {
	prepareTests("Distances");

	makeTest( "Texture binding", test01() );

	makeTest( "Distances calculation", test02() );

	endTests;
}
//----------------------------------------------------------------------------

bool test01 () {
	return testRawTextures();
}
//----------------------------------------------------------------------------

bool test02 () {
	return testDistanceCalculation();
}
//----------------------------------------------------------------------------
//----------------------------------------------------------------------------
